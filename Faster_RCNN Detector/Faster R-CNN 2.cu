#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hipblas.h>
#include <cstring>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <time.h>
#include <chrono>

#define IMAGE_HEIGHT 28
#define IMAGE_WIDTH 28
#define NUM_CHANNELS 1
#define NUM_CLASSES 10
#define BATCH_SIZE 128

using namespace std;
using namespace std::chrono;

// Function to convert h5 weights to arrays
void convert_weights_to_arrays(const string& weight_file, float** weights, int& num_weights) 
{
    ifstream file(weight_file, ios::binary | ios::ate);
    streamsize size = file.tellg();
    file.seekg(0, ios::beg);
    vector<char> buffer(size);
    if (!file.read(buffer.data(), size)) {
        cerr << "Error reading weight file" << endl;
    }
    num_weights = size / sizeof(float);
    *weights = new float[num_weights];
    memcpy(*weights, buffer.data(), size);
}

// Function to perform forward pass on the GPU
void forward_pass(const float* input, float* output, float** weights, const int& num_weights, hipdnnHandle_t& cudnn, hipblasHandle_t& cublas) {
    // Initialize tensors
    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CHANNELS, IMAGE_HEIGHT, IMAGE_WIDTH);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CLASSES, 1, 1);

    // Allocate memory on GPU
    float* gpu_input;
    float* gpu_output;
    hipMalloc((void**)&gpu_input, BATCH_SIZE * NUM_CHANNELS * IMAGE_HEIGHT * IMAGE_WIDTH * sizeof(float));
    hipMalloc((void**)&gpu_output, BATCH_SIZE * NUM_CLASSES * sizeof(float));

    // Copy input data to GPU
    hipMemcpy(gpu_input, input, BATCH_SIZE * NUM_CHANNELS * IMAGE_HEIGHT * IMAGE_WIDTH * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward pass on GPU
    // Create a linear layer on GPU
    hipdnnFilterDescriptor_t weight, bias_desc;
    hipdnnCreateFilterDescriptor(&weight);
    hipdnnCreateFilterDescriptor(&bias_desc);
    hipdnnSetFilter4dDescriptor(weight, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, NUM_CHANNELS, NUM_CLASSES, IMAGE_HEIGHT, IMAGE_WIDTH);
    hipdnnSetFilter4dDescriptor(bias_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, NUM_CLASSES, 1, 1, 1);

    // Allocate memory on GPU for weights and biases
    float* gpu_weights;
    float* gpu_biases;
    hipMalloc((void**)&gpu_weights, num_weights * sizeof(float));
    hipMalloc((void**)&gpu_biases, NUM_CLASSES * sizeof(float));

    // Copy weights and biases to GPU
    hipMemcpy(gpu_weights, *weights, num_weights * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_biases, *weights + num_weights - NUM_CLASSES, NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

    // Perform convolution
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn, &alpha, input_desc, gpu_input, weight, gpu_weights, HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, &beta, output_desc, gpu_output);

    // Perform bias addition
    hipblasSgemm(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, NUM_CLASSES, BATCH_SIZE, 1, &alpha, gpu_biases, NUM_CLASSES, gpu_output, NUM_CLASSES, &beta, gpu_output, NUM_CLASSES);

    // Copy output data back to host
    hipMemcpy(output, gpu_output, BATCH_SIZE * NUM_CLASSES * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(weight);
    hipdnnDestroyFilterDescriptor(bias_desc);
    hipFree(gpu_input);
    hipFree(gpu_output);
    hipFree(gpu_weights);
    hipFree(gpu_biases);

}

int main(int argc, char** argv) 
{
    if (argc != 2) 
    {
        cerr << "Usage: " << argv[0] << " <weight file>" << endl;
        return 1;
    }

    // Load weights
    float* weights;
    int num_weights;
    convert_weights_to_arrays(argv[1], &weights, num_weights);

    // Initialize CUDA and cuDNN
    hipSetDevice(0);
    hipdnnHandle_t cudnn;

    hipblasHandle_t cublas;
    hipdnnCreate(&cudnn);
    hipblasCreate(&cublas);

    // Initialize input and output arrays
    float* input = new float[BATCH_SIZE * NUM_CHANNELS * IMAGE_HEIGHT * IMAGE_WIDTH];
    float* output = new float[BATCH_SIZE * NUM_CLASSES];

    // Perform image classification
    classify_images(cudnn, cublas, weights, num_weights, input, output);

    // Clean up
    hipdnnDestroy(cudnn);
    hipblasDestroy(cublas);
    delete[] input;
    delete[] output;
    delete[] weights;

    return 0;

}
    
    
